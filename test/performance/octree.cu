/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Benchmark cornerstone octree generation on the GPU
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <iostream>

#include <thrust/reduce.h>

#include "cstone/halos/discovery.cuh"
#include "cstone/tree/octree.cuh"

#include "coord_samples/random.hpp"

using namespace cstone;

int main()
{
    using KeyType = unsigned;
    Box<double> box{-1, 1};

    unsigned numParticles = 2000000;
    unsigned bucketSize = 16;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    RandomGaussianCoordinates<double, KeyType> randomBox(numParticles, box);

    thrust::device_vector<KeyType> tree    = std::vector<KeyType>{0, nodeRange<KeyType>(0)};
    thrust::device_vector<unsigned> counts = std::vector<unsigned>{numParticles};

    thrust::device_vector<KeyType>       tmpTree;
    thrust::device_vector<TreeNodeIndex> workArray;

    thrust::device_vector<KeyType> particleCodes(randomBox.particleKeys().begin(), randomBox.particleKeys().end());

    hipEventRecord(start, hipStreamDefault);

    while(!updateOctreeGpu(thrust::raw_pointer_cast(particleCodes.data()),
                           thrust::raw_pointer_cast(particleCodes.data() + numParticles),
                           bucketSize, tree, counts, tmpTree, workArray));

    hipEventRecord(stop, hipStreamDefault);
    hipEventSynchronize(stop);

    float t0;
    hipEventElapsedTime(&t0, start, stop);
    std::cout << "build time from scratch " << t0/1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(counts.begin(), counts.end(), 0) << std::endl;

    hipEventRecord(start, hipStreamDefault);

    updateOctreeGpu(thrust::raw_pointer_cast(particleCodes.data()),
                    thrust::raw_pointer_cast(particleCodes.data() + numParticles),
                    bucketSize, tree, counts, tmpTree, workArray);

    hipEventRecord(stop, hipStreamDefault);
    hipEventSynchronize(stop);

    float t1;
    hipEventElapsedTime(&t1, start, stop);
    std::cout << "build time with guess " << t1/1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(counts.begin(), counts.end(), 0) << std::endl;

    // halo discovery benchmark

    thrust::device_vector<BinaryNode<KeyType>> binaryTree(nNodes(tree));
    createBinaryTreeGpu(thrust::raw_pointer_cast(tree.data()), nNodes(tree),
                        thrust::raw_pointer_cast(binaryTree.data()));

    thrust::device_vector<float> haloRadii(nNodes(tree), 0.01);
    thrust::device_vector<int>   flags(nNodes(tree), 0);

    hipEventRecord(start, hipStreamDefault);
    findHalosGpu(thrust::raw_pointer_cast(tree.data()),
                 thrust::raw_pointer_cast(binaryTree.data()),
                 thrust::raw_pointer_cast(haloRadii.data()),
                 box, 0, nNodes(tree) / 4,
                 thrust::raw_pointer_cast(flags.data())
                 );

    hipEventRecord(stop, hipStreamDefault);
    hipEventSynchronize(stop);

    float t2;
    hipEventElapsedTime(&t2, start, stop);
    std::cout << "halo discovery " << t2/1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(flags.begin(), flags.end(), 0) << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
