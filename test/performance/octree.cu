/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Benchmark cornerstone octree generation on the GPU
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <iostream>

#include <thrust/reduce.h>

#include "cstone/halos/discovery.cuh"
#include "cstone/tree/octree.cuh"

#include "coord_samples/random.hpp"

#include "timing.cuh"

using namespace cstone;

int main()
{
    using KeyType = unsigned;
    Box<double> box{-1, 1};

    unsigned numParticles = 2000000;
    unsigned bucketSize   = 16;

    RandomGaussianCoordinates<double, MortonKey<KeyType>> randomBox(numParticles, box);

    thrust::device_vector<KeyType> tree    = std::vector<KeyType>{0, nodeRange<KeyType>(0)};
    thrust::device_vector<unsigned> counts = std::vector<unsigned>{numParticles};

    thrust::device_vector<KeyType>       tmpTree;
    thrust::device_vector<TreeNodeIndex> workArray;

    thrust::device_vector<KeyType> particleCodes(randomBox.particleKeys().begin(), randomBox.particleKeys().end());

    auto fullBuild = [&]()
    {
        while(!updateOctreeGpu(thrust::raw_pointer_cast(particleCodes.data()),
                               thrust::raw_pointer_cast(particleCodes.data() + numParticles),
                               bucketSize, tree, counts, tmpTree, workArray));
    };

    float buildTime = timeGpu(fullBuild);
    std::cout << "build time from scratch " << buildTime / 1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(counts.begin(), counts.end(), 0) << std::endl;

    auto updateTree = [&]()
    {
        updateOctreeGpu(thrust::raw_pointer_cast(particleCodes.data()),
                        thrust::raw_pointer_cast(particleCodes.data() + numParticles),
                        bucketSize, tree, counts, tmpTree, workArray);
    };

    float updateTime = timeGpu(updateTree);
    std::cout << "build time with guess " << updateTime / 1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(counts.begin(), counts.end(), 0) << std::endl;

    // halo discovery benchmark

    thrust::device_vector<BinaryNode<KeyType>> binaryTree(nNodes(tree));
    createBinaryTreeGpu(thrust::raw_pointer_cast(tree.data()), nNodes(tree),
                        thrust::raw_pointer_cast(binaryTree.data()));

    thrust::device_vector<float> haloRadii(nNodes(tree), 0.01);
    thrust::device_vector<int>   flags(nNodes(tree), 0);

    auto findHalosLambda = [&]()
    {
        findHalosGpu(thrust::raw_pointer_cast(tree.data()),
                     thrust::raw_pointer_cast(binaryTree.data()),
                     thrust::raw_pointer_cast(haloRadii.data()),
                     box, 0, nNodes(tree) / 4,
                     thrust::raw_pointer_cast(flags.data()));
    };

    float findTime = timeGpu(findHalosLambda);
    std::cout << "halo discovery " << findTime / 1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(flags.begin(), flags.end(), 0) << std::endl;
}
