#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief  Find neighbors in Morton code sorted x,y,z arrays
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "findneighbors.cuh"

template<class T, class Integer>
__global__ void findNeighborsKernel(const T* x, const T* y, const T* z, const T* h, int firstId, int lastId, int n,
                                    cstone::Box<T> box, const Integer* particleKeys,
                                    int* neighbors, int* neighborsCount, int ngmax)
{
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned id = firstId + tid;
    if (id < lastId)
    {
        cstone::findNeighbors(id, x, y, z, h, box, particleKeys, neighbors + tid*ngmax, neighborsCount + tid, n, ngmax);
    }
}

template<class T, class Integer>
void findNeighborsGpu(const T* x, const T* y, const T* z, const T* h, int firstId, int lastId, int n,
                       cstone::Box<T> box, const Integer* particleKeys, int* neighbors, int* neighborsCount, int ngmax,
                       hipStream_t stream)
{
    unsigned numThreads = 256;
    unsigned numBlocks  = iceil(n, numThreads);
    findNeighborsKernel<<<numBlocks, numThreads, 0, stream>>>
        (x, y, z, h, firstId, lastId, n, box, particleKeys, neighbors, neighborsCount, ngmax);
}

template FIND_NEIGHBORS_GPU(float,  uint32_t)
template FIND_NEIGHBORS_GPU(float,  uint64_t)
template FIND_NEIGHBORS_GPU(double, uint32_t)
template FIND_NEIGHBORS_GPU(double, uint64_t)
