#include "hip/hip_runtime.h"

/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief  Utility for GPU-direct halo particle exchange
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/primitives/math.hpp"
#include "cstone/primitives/stl.hpp"
#include "cstone/util/array.hpp"
#include "gather_halos_gpu.h"

namespace cstone
{

template<class T, class IndexType>
__global__ void gatherRangesKernel(const IndexType* rangeScan,
                                   const IndexType* rangeOffsets,
                                   int numRanges,
                                   const T* src,
                                   T* buffer,
                                   size_t bufferSize)
{
    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < bufferSize)
    {
        IndexType rangeIdx = stl::upper_bound(rangeScan, rangeScan + numRanges, tid) - rangeScan - 1;

        IndexType srcIdx = rangeOffsets[rangeIdx] + tid - rangeScan[rangeIdx];
        buffer[tid]      = src[srcIdx];
    }
}

template<class T, class IndexType>
void gatherRanges(const IndexType* rangeScan,
                  const IndexType* rangeOffsets,
                  int numRanges,
                  const T* src,
                  T* buffer,
                  size_t bufferSize)
{
    int numThreads = 256;
    int numBlocks  = iceil(bufferSize, numThreads);
    gatherRangesKernel<<<numBlocks, numThreads>>>(rangeScan, rangeOffsets, numRanges, src, buffer, bufferSize);
}

template void gatherRanges(const unsigned*, const unsigned*, int, const int*, int*, size_t);
template void gatherRanges(const uint64_t*, const uint64_t*, int, const int*, int*, size_t);

template void
gatherRanges(const unsigned*, const unsigned*, int, const util::array<float, 1>*, util::array<float, 1>*, size_t);
template void
gatherRanges(const unsigned*, const unsigned*, int, const util::array<float, 2>*, util::array<float, 2>*, size_t);
template void
gatherRanges(const unsigned*, const unsigned*, int, const util::array<float, 3>*, util::array<float, 3>*, size_t);
template void
gatherRanges(const unsigned*, const unsigned*, int, const util::array<float, 4>*, util::array<float, 4>*, size_t);
template void

gatherRanges(const uint64_t*, const uint64_t*, int, const util::array<float, 1>*, util::array<float, 1>*, size_t);
template void
gatherRanges(const uint64_t*, const uint64_t*, int, const util::array<float, 2>*, util::array<float, 2>*, size_t);
template void
gatherRanges(const uint64_t*, const uint64_t*, int, const util::array<float, 3>*, util::array<float, 3>*, size_t);
template void
gatherRanges(const uint64_t*, const uint64_t*, int, const util::array<float, 4>*, util::array<float, 4>*, size_t);

template<class T, class IndexType>
__global__ void scatterRangesKernel(const IndexType* rangeScan,
                                    const IndexType* rangeOffsets,
                                    int numRanges,
                                    T* dest,
                                    const T* buffer,
                                    size_t bufferSize)
{
    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < bufferSize)
    {
        IndexType rangeIdx = stl::upper_bound(rangeScan, rangeScan + numRanges, tid) - rangeScan - 1;

        IndexType destIdx = rangeOffsets[rangeIdx] + tid - rangeScan[rangeIdx];
        dest[destIdx]     = buffer[tid];
    }
}

template<class T, class IndexType>
void scatterRanges(const IndexType* rangeScan,
                   const IndexType* rangeOffsets,
                   int numRanges,
                   T* dest,
                   const T* buffer,
                   size_t bufferSize)
{
    int numThreads = 256;
    int numBlocks  = iceil(bufferSize, numThreads);
    scatterRangesKernel<<<numBlocks, numThreads>>>(rangeScan, rangeOffsets, numRanges, dest, buffer, bufferSize);
}

template void scatterRanges(const unsigned*, const unsigned*, int, uint8_t*, const uint8_t*, size_t);
template void scatterRanges(const unsigned*, const unsigned*, int, int*, const int*, size_t);
template void scatterRanges(const uint64_t*, const uint64_t*, int, int*, const int*, size_t);

template void
scatterRanges(const unsigned*, const unsigned*, int, util::array<float, 1>*, const util::array<float, 1>*, size_t);
template void
scatterRanges(const unsigned*, const unsigned*, int, util::array<float, 2>*, const util::array<float, 2>*, size_t);
template void
scatterRanges(const unsigned*, const unsigned*, int, util::array<float, 3>*, const util::array<float, 3>*, size_t);
template void
scatterRanges(const unsigned*, const unsigned*, int, util::array<float, 4>*, const util::array<float, 4>*, size_t);

} // namespace cstone
